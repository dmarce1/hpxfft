#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hpxfft/cuda_check.hpp>
#include <vector>
#include <complex>

__global__ void fft_kernel_step1(float *cosi, float *sine, int N) {
	for (int i = threadIdx.x; i < N / 2; i += blockDim.x) {
		if (i < N / 2) {
			const float omega = -2.0 * M_PI * i / N;
			cosi[i] = std::cos(omega);
			sine[i] = std::sin(omega);
		}
	}

}

__global__ void fft_kernel_step2(float *Aptr, float *Bptr, float *cosi, float *sine, int N) {
	int level = 0;
	for (auto i = N; i > 1; i >>= 1) {
		level++;
	}
	if ((1 << level) != N) {
		if (threadIdx.x == 0) {
			printf("FFT requires power of two!\n");
		}
	}

	auto* A = Aptr + blockIdx.x * N;
	auto* B = Bptr + blockIdx.x * N;

	for (auto i = threadIdx.x; i < N; i += blockDim.x) {
		auto j = 0;
		int l = i;
		for (int k = 0; k < level; k++) {
			j = (j << 1) | (l & 1);
			l >>= 1;
		}
		if (j > i) {
			float tmp = A[i];
			A[i] = A[j];
			A[j] = tmp;
			tmp = B[i];
			B[i] = B[j];
			B[j] = tmp;
		}
	}

	for (int P = 2; P <= N; P *= 2) {
		const int s = N / P;
		if (N / P <= blockDim.x) {
			const int imax = ((N - 1) / blockDim.x + 1) * blockDim.x;
			for (int i = threadIdx.x * P; i < imax; i += blockDim.x * P) {
				if (i < N) {
					int k = 0;
					for (int j = i; j < i + P / 2; j++) {
						const auto treal = A[j + P / 2] * cosi[k] - B[j + P / 2] * sine[k];
						const auto timag = A[j + P / 2] * sine[k] + B[j + P / 2] * cosi[k];
						A[j + P / 2] = A[j] - treal;
						B[j + P / 2] = B[j] - timag;
						A[j] += treal;
						B[j] += timag;
						k += s;
					}
				}
				__syncthreads();
			}
		} else {
			for (int i = 0; i < N; i += P) {
				int k = threadIdx.x * s;
				const int jmax = ((P / 2 - 1) / blockDim.x + 1) * blockDim.x + i;
				for (int j = i + threadIdx.x; j < jmax; j += blockDim.x) {
					if (j < i + P / 2) {
						const auto treal = A[j + P / 2] * cosi[k] - B[j + P / 2] * sine[k];
						const auto timag = A[j + P / 2] * sine[k] + B[j + P / 2] * cosi[k];
						A[j + P / 2] = A[j] - treal;
						B[j + P / 2] = B[j] - timag;
						A[j] += treal;
						B[j] += timag;
						k += blockDim.x * s;
					}
					__syncthreads();
				}
			}
		}
	}
}

void fft_cuda(std::vector<std::vector<std::complex<float>> > &X) {
	float *A;
	float *B;
	float *A_dev;
	float *B_dev;
	float *cosi_dev;
	float *sine_dev;
	const int cnt = X.size();
	const int N = X[0].size();
	const int size = N * sizeof(float);
	CUDA_CHECK(hipHostMalloc((void**) &A, cnt * size));
	CUDA_CHECK(hipHostMalloc((void**) &B, cnt * size));
	CUDA_CHECK(hipMalloc((void**) &A_dev, cnt * size));
	CUDA_CHECK(hipMalloc((void**) &B_dev, cnt * size));
	CUDA_CHECK(hipMalloc((void**) &cosi_dev, size));
	CUDA_CHECK(hipMalloc((void**) &sine_dev, size));
	for (int j = 0; j < cnt; j++) {
		for (int i = 0; i < N; i++) {
			A[i + j * N] = X[j][i].real();
			B[i + j * N] = X[j][i].imag();
		}
	}
	fft_kernel_step1<<<1,32>>>(cosi_dev,sine_dev,N);
	hipMemcpy(A_dev, A, cnt * size, hipMemcpyHostToDevice);
	hipMemcpy(B_dev, B, cnt * size, hipMemcpyHostToDevice);
	fft_kernel_step2<<<cnt,32>>>(A_dev, B_dev, cosi_dev,sine_dev,N);
	CUDA_CHECK(hipMemcpy(A, A_dev, cnt * size, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(B, B_dev, cnt * size, hipMemcpyDeviceToHost));
	for (int j = 0; j < cnt; j++) {
		for (int i = 0; i < N; i++) {
			reinterpret_cast<float (&)[2]>(X[j][i])[0] = A[i + j * N];
			reinterpret_cast<float (&)[2]>(X[j][i])[1] = B[i + j * N];
		}
	}
	CUDA_CHECK(hipHostFree(A));
	CUDA_CHECK(hipHostFree(B));
	CUDA_CHECK(hipFree(A_dev));
	CUDA_CHECK(hipFree(B_dev));
	CUDA_CHECK(hipFree(cosi_dev));
	CUDA_CHECK(hipFree(sine_dev));
}

